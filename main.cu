#include <cstdio>
#include "hipcub/hipcub.hpp"

#define cudaCheckError() {                                          \
  hipError_t e=hipGetLastError();                                  \
  if(e!=hipSuccess) {                                               \
  printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
  exit(0); \
  }                                                                  \
}

void RunTestCub(char* label, int N, int REPEAT, int* src, int checksum) {
  int *in, *out;
  hipEvent_t start,stop;
  
  hipMalloc(&in,sizeof(int)*N);
  hipMalloc(&out,sizeof(int)*1024);  //only stable version needs multiple elements, all others only need 1
  hipEventCreate(&start);
  hipEventCreate(&stop);
  cudaCheckError();

  hipMemcpy(in,src,N*sizeof(int),hipMemcpyHostToDevice);

  size_t temp_storage_bytes;
  void* temp_storage=NULL;
  hipcub::DeviceReduce::Reduce(temp_storage, temp_storage_bytes, in, out, N, hipcub::Sum(), 0);
  hipMalloc(&temp_storage,temp_storage_bytes);

  hipDeviceSynchronize();
  cudaCheckError();
  hipEventRecord(start);

  for(int i=0;i<REPEAT;i++) {
    hipcub::DeviceReduce::Reduce(temp_storage, temp_storage_bytes, in, out, N, hipcub::Sum(), 0);
  }
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();

  float time_ms;
  hipEventElapsedTime(&time_ms,start,stop);
  float time_s=time_ms/(float)1e3;

  float GB=(float)N*sizeof(int)*REPEAT;
  float GBs=GB/time_s/(float)1e9;

  int sum;
  hipMemcpy(&sum,out,sizeof(int),hipMemcpyDeviceToHost);
  cudaCheckError();

  char *valid;
  if(sum==checksum) 
    valid="CORRECT";
  else
    valid="INCORRECT";

  printf("%s: %s, Time: %f s, GB/s: %f\n", label, valid, time_s, GBs); 
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(in);
  hipFree(out);
  hipFree(temp_storage);
  cudaCheckError();
}


int main(int argc, char** argv)
{
  if(argc!=3) {
    printf("Usage: ./reduce num_elems repeat\n");
    exit(0);
  }
  int NUM_ELEMS=atoi(argv[1]);
  int REPEAT=atoi(argv[2]);

  printf("NUM_ELEMS: %d, REPEAT: %d\n", NUM_ELEMS, REPEAT);

  int* vals=(int*)malloc(NUM_ELEMS*sizeof(int));
  int checksum =0;
  for(int i=0;i<NUM_ELEMS;i++) {
    vals[i]=rand()%4;
    checksum+=vals[i];
  }

  RunTestCub("device_reduce_cub",NUM_ELEMS,REPEAT,vals,checksum);
  
  free(vals);

}
